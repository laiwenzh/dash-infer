
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) Alibaba, Inc. and its affiliates.
 * @file    bf16_simt.cu
 */

#ifdef ENABLE_BF16
#include "attn/qkv/impl_simt.cuh"
#include "common/data_type.h"

namespace span {
template struct QKVWorkspaceBytes<SaArch::SIMT, bfloat16_t>;
template struct QKVLauncher<SaArch::SIMT, bfloat16_t>;
}  // namespace span
#endif
